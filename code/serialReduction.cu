﻿// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>


#define TILE_WIDTH 16
void reductionSerial(int *A, int *B, int n)
{
	int res = 0;
	for (int i = 0; i < n; ++i)
	{
		res += A[i];
	}
	B[0] = res;
}

void randomInit(int *data, int size)
{
	srand(time(NULL));
	
	for (int i = 0; i < size; ++i)
	{
		data[i] = rand() % 10;
	}
}

/**
* Run a simple test of matrix multiplication using CUDA
*/
int Reduction(int argc, char **argv, int n)
{
	// Allocate host memory for array
	unsigned int size_A = n;
	unsigned int mem_size_A = sizeof(int)* size_A;
	int *h_A = (int *)malloc(mem_size_A);
	unsigned int size_B = n;
	unsigned int mem_size_B = sizeof(int)* size_B;
	int *h_B = (int *)malloc(mem_size_B);

	// Initialize Array
	randomInit(h_A, size_A);


	hipError_t error;

	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	reductionSerial(h_A, h_B, n);

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	printf("Elapsed time in msec = %f\n", msecTotal);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	


	// Clean up memory
	free(h_A);
	free(h_B);

	return EXIT_SUCCESS;
}


/**
* Program main
*/
int main(int argc, char **argv)
{
	printf("[Reduction Using CUDA] - Starting...\n");

	// By default, we use device 0
	int devID = 0;
	hipSetDevice(devID);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}

	// Size of square matrices
	size_t n = 0;
	printf("[-] N = ");
	scanf("%u", &n);

	printf("Array size is (%d, %d)\n", n, 1);

	int reduction_result = Reduction(argc, argv, n);

	exit(reduction_result);
}
